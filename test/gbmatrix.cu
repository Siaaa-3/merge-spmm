#define GRB_USE_APSPIE
//#define private public

#include <iostream>
#include <algorithm>
#include <string>

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime_api.h>

#include "graphblas/graphblas.hpp"

#include <boost/program_options.hpp>
#include <test/test.hpp>

int main( int argc, char** argv )
{
  std::vector<graphblas::Index> row_indices;
  std::vector<graphblas::Index> col_indices;
  std::vector<float> values;
  graphblas::Index nrows, ncols, nvals;

  // Parse arguments
  namespace po = boost::program_options;
  po::variables_map vm;
  parseArgs( argc, argv, vm );
  bool DEBUG = false;

  // Read in sparse matrix
  if (argc < 2) {
    fprintf(stderr, "Usage: %s [matrix-market-filename]\n", argv[0]);
    exit(1);
  } else { 
    readMtx( argv[argc-1], row_indices, col_indices, values, nrows, ncols, 
    nvals, DEBUG );
  }

  // Matrix A
  graphblas::Matrix<float> a(nrows, ncols);
  a.build( row_indices, col_indices, values, nvals );
  a.nrows( nrows );
  a.ncols( ncols );
  a.nvals( nvals );
  if( DEBUG ) a.print();

  a.extractTuples( row_indices, col_indices, values );

  for( int i=0; i<nvals; i++ )
    std::cout << i << " " << row_indices[i] << " " << col_indices[i] << " " <<
        values[i] << std::endl;
}
